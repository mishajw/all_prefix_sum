
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#define CUDA_ERROR(err, message) \
  do { \
    hipError_t err2 = err; \
    if (err2 != hipSuccess) { \
      fprintf(stderr, "%s: %s\n", message, hipGetErrorString(err2)); \
      exit(1); \
    } \
  } while (0);

#define GLOBAL_INDEX blockIdx.x * blockDim.x + threadIdx.x

#define BLOCK_SIZE 1024

// The size of the array to test on
static const size_t ARRAY_SIZE = 1000;

typedef int32_t num_t;

// Performs exclusive Blelloch scan on a block level
__global__
void blelloch_block_scan(const num_t *g_input, num_t *g_output, size_t length) {
  __shared__ num_t s_temp[BLOCK_SIZE * 2];
  size_t global_index = GLOBAL_INDEX;
  size_t index = threadIdx.x;
  size_t offset = 1;

  // Copy global memory into shared
  if (global_index * 2 < length) {
    s_temp[index * 2] = g_input[global_index * 2];
  }
  if (global_index * 2 + 1 < length) {
    s_temp[index * 2 + 1] = g_input[global_index * 2 + 1];
  }

  // Up sweep
  for (size_t d = BLOCK_SIZE; d > 0; d /= 2) {
    __syncthreads();

    if (index < d) {
      size_t a = offset * (2 * index + 1) - 1;
      size_t b = offset * (2 * index + 2) - 1;
      s_temp[b] += s_temp[a];
    }

    offset *= 2;
  }

  // Reset last element
  if (index == 0) {
    s_temp[BLOCK_SIZE * 2 - 1] = 0;
  }

  // Down sweep
  for (size_t d = 1; d < BLOCK_SIZE * 2; d *= 2) {
    offset /= 2;
    __syncthreads();
    if (index < d) {
      size_t a = offset * (2 * index + 1) - 1;
      size_t b = offset * (2 * index + 2) - 1;
      num_t t = s_temp[a];
      s_temp[a] = s_temp[b];
      s_temp[b] += t;
    }
  }

  __syncthreads();

  // Copy results into global memory
  if (global_index * 2 < length) {
    g_output[global_index * 2] = s_temp[index * 2];
  }
  if (global_index * 2 + 1 < length) {
    g_output[global_index * 2 + 1] = s_temp[index * 2 + 1];
  }
}

// Copy the _inclusive_ ends of block scans into an array
// Must be inclusive, as they are added to all array elements afterwards. But
// to make it inclusive, we need the original array
// TODO: Each thread makes three global memory accesses - is there any way we
// can avoid this?
__global__
void copy_block_scan_ends(
    const num_t *original_input,
    const num_t *g_input,
    num_t *g_output,
    const size_t length) {

  size_t global_index = GLOBAL_INDEX;

  // Each thread will process two items of data so that we can process more
  // before needing a level (n + 1) scan
  // TODO: Is this the correct decision? We sacrifice some parallelism for not
  // needing level 3 block scan, but why only 2x and not more?

  size_t i1 = (global_index + 1) * BLOCK_SIZE * 2 - 1;
  if (i1 < length) {
    g_output[global_index] = g_input[i1] + original_input[i1];
  }

  size_t i2 = (BLOCK_SIZE * BLOCK_SIZE * 2) + i1;
  if (i2 < length) {
    g_output[global_index + BLOCK_SIZE] = g_input[i2] + original_input[i2];
  }
}

// Add the block scan ends back onto the original array
__global__
void add_block_scan_ends(
    num_t *g_input, const num_t *g_block_ends, size_t length) {

  size_t global_index = GLOBAL_INDEX;
  if (global_index < length) {
    g_input[global_index] += g_block_ends[global_index / (BLOCK_SIZE * 2)];
  }
}

// Performs all prefix sum on `input` and stores the result in `output` in
// parallel on a GPU
// Assumes both `input` and `output` are allocated with size `length`
void scan(const num_t *input, num_t *output, size_t length) {
  hipError_t err;
  size_t array_size = sizeof(num_t) * length;

  // Set up input on device
  num_t *g_input = NULL;
  err = hipMalloc((void **)&g_input, array_size);
  CUDA_ERROR(err, "Couldn't allocate memory for input on device");
  err = hipMemcpy(g_input, input, array_size, hipMemcpyHostToDevice);
  CUDA_ERROR(err, "Couldn't copy input to device");

  // Setup output on device
  num_t *g_output = NULL;
  err = hipMalloc((void **)&g_output, array_size);
  CUDA_ERROR(err, "Couldn't allocate memory for output on device");

  if (length <= BLOCK_SIZE * 2) {
    blelloch_block_scan<<<1, BLOCK_SIZE>>>(g_input, g_output, length);
    CUDA_ERROR(hipGetLastError(), "Couldn't perform block scan");
  } else if (length <= BLOCK_SIZE * BLOCK_SIZE * 4) {
    // Perform block scan on individual blocks of the input
    size_t num_blocks = 1 + (length - 1) / BLOCK_SIZE;
    blelloch_block_scan<<<num_blocks, BLOCK_SIZE>>>(g_input, g_output, length);
    CUDA_ERROR(hipGetLastError(), "Couldn't perform block scan");

    // Create array for block ends
    num_t *g_block_scan_ends = NULL;
    CUDA_ERROR(
        hipMalloc((void**)&g_block_scan_ends, sizeof(num_t) * num_blocks),
        "Couldn't allocated memory for scan_ends");

    // Fill block scan ends
    size_t ends_num_blocks = 1 + (length - 1) / (BLOCK_SIZE * BLOCK_SIZE);
    copy_block_scan_ends<<<ends_num_blocks, BLOCK_SIZE>>>(
        g_input, g_output, g_block_scan_ends, length);
    CUDA_ERROR(hipGetLastError(), "Couldn't get block scan ends");

    // Perform prefix sum of block scan ends
    blelloch_block_scan<<<ends_num_blocks, BLOCK_SIZE>>>(
        g_block_scan_ends, g_block_scan_ends, num_blocks);
    CUDA_ERROR(
        hipGetLastError(), "Couldn't perform block scan on block scan ends");

    // Add the block ends to the output
    add_block_scan_ends<<<num_blocks, BLOCK_SIZE>>>(
        g_output, g_block_scan_ends, length);
    CUDA_ERROR(hipGetLastError(), "Couldn't add block scan ends");
  } else {
    // TODO: Implement
  }

  // Copy results to host
  err = hipMemcpy(output, g_output, array_size, hipMemcpyDeviceToHost);
  CUDA_ERROR(err, "Couldn't copy output to host");
}

// Performs all prefix sum on `input` and stores the result in `output`
// sequentially on the CPU
// Assumes both `input` and `output` are allocated with size `length`
void sequential_scan(const num_t *input, num_t *output, size_t length) {
  for (size_t i = 1; i < length; i++) {
    output[i] = output[i - 1] + input[i - 1];
  }
}

// Fills the array `array` with `length` random values from 0-9 inclusive
void fill_random_array(num_t *array, size_t length) {
  for (size_t i = 0; i < length; i++) {
    array[i] = rand() % 10;
  }
}

// Print how two arrays `a` and `b` differ, up to some length `length`
// Returns true if the arrays are equal, and false otherwise
bool print_array_equality(num_t *a, num_t *b, size_t length) {
  bool are_equal = true;

  for (size_t i = 0; i < length; i++) {
    if (a[i] != b[i]) {
      are_equal = false;
      printf(
          "Arrays differ at index %ld, with values %d and %d\n", i, a[i], b[i]);
    }
  }

  return are_equal;
}

int main() {
  // Set up the input to scan
  num_t *input = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);
  fill_random_array(input, ARRAY_SIZE);

  // Set up the output for sequential function, to be used as ground truth for
  // comparison
  num_t *truth_output = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);

  // Set up the output for the parallel function
  num_t *output = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);

  // Run the sequential scan
  // TODO: Time this operation
  sequential_scan(input, truth_output, ARRAY_SIZE);

  // Run the parallel scan
  // TODO: Time this operation
  scan(input, output, ARRAY_SIZE);

  // Compare solutions
  bool are_equal = print_array_equality(truth_output, output, ARRAY_SIZE);

  if (are_equal) {
    printf("Success!\n");
  }

  return are_equal ? 0 : 1;
}

