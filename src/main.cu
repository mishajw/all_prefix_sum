
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#define CUDA_ERROR(err, message) \
  do { \
    if (err != hipSuccess) { \
      fprintf(stderr, "%s: %s\n", message, hipGetErrorString(err)); \
      exit(1); \
    } \
  } while (0);

#define GLOBAL_INDEX blockIdx.x * blockDim.x + threadIdx.x

#define BLOCK_SIZE 1024

// The size of the array to test on
static const size_t ARRAY_SIZE = 1000;

typedef int32_t num_t;

// Performs exclusive Blelloch scan on a block level
__global__
void blelloch_block_scan(const num_t *g_input, num_t *g_output, size_t length) {
  __shared__ num_t s_temp[BLOCK_SIZE * 2];
  size_t global_index = GLOBAL_INDEX;
  size_t index = threadIdx.x;
  size_t offset = 1;

  // Copy global memory into shared
  if (global_index * 2 < length) {
    s_temp[index * 2] = g_input[global_index * 2];
  }
  if (global_index * 2 + 1 < length) {
    s_temp[index * 2 + 1] = g_input[global_index * 2 + 1];
  }

  // Up sweep
  for (size_t d = BLOCK_SIZE; d > 0; d /= 2) {
    __syncthreads();

    if (global_index < d) {
      size_t a = offset * (2 * index + 1) - 1;
      size_t b = offset * (2 * index + 2) - 1;
      s_temp[b] += s_temp[a];
    }

    offset *= 2;
  }

  // Reset last element
  if (index == 0) {
    s_temp[BLOCK_SIZE * 2 - 1] = 0;
  }

  // Down sweep
  for (size_t d = 1; d < BLOCK_SIZE * 2; d *= 2) {
    offset /= 2;
    __syncthreads();
    if (global_index < d) {
      size_t a = offset * (2 * index + 1) - 1;
      size_t b = offset * (2 * index + 2) - 1;
      num_t t = s_temp[a];
      s_temp[a] = s_temp[b];
      s_temp[b] += t;
    }
  }

  __syncthreads();

  // Copy results into global memory
  if (global_index * 2 < length) {
    g_output[global_index * 2] = s_temp[index * 2];
  }
  if (global_index * 2 + 1 < length) {
    g_output[global_index * 2 + 1] = s_temp[index * 2 + 1];
  }
}

// Performs all prefix sum on `input` and stores the result in `output` in
// parallel on a GPU
// Assumes both `input` and `output` are allocated with size `length`
void scan(const num_t *input, num_t *output, size_t length) {
  hipError_t err;
  size_t array_size = sizeof(num_t) * length;

  // Set up input on device
  num_t *g_input = NULL;
  err = hipMalloc((void **)&g_input, array_size);
  CUDA_ERROR(err, "Couldn't allocate memory for input on device");
  err = hipMemcpy(g_input, input, array_size, hipMemcpyHostToDevice);
  CUDA_ERROR(err, "Couldn't copy input to device");

  // Setup output on device
  num_t *g_output = NULL;
  err = hipMalloc((void **)&g_output, array_size);
  CUDA_ERROR(err, "Couldn't allocate memory for output on device");

  if (length <= BLOCK_SIZE) {
    blelloch_block_scan<<<1, BLOCK_SIZE>>>(g_input, g_output, length);
    CUDA_ERROR(hipGetLastError(), "Couldn't perform block scan");
  } else {
    // TODO: Implement
  }

  // TODO: Check if necessary
  hipDeviceSynchronize();

  // Copy results to host
  err = hipMemcpy(output, g_output, array_size, hipMemcpyDeviceToHost);
  CUDA_ERROR(err, "Couldn't copy output to host");
}

// Performs all prefix sum on `input` and stores the result in `output`
// sequentially on the CPU
// Assumes both `input` and `output` are allocated with size `length`
void sequential_scan(const num_t *input, num_t *output, size_t length) {
  for (size_t i = 1; i < length; i++) {
    output[i] = output[i - 1] + input[i - 1];
  }
}

// Fills the array `array` with `length` random values from 0-9 inclusive
void fill_random_array(num_t *array, size_t length) {
  for (size_t i = 0; i < length; i++) {
    array[i] = rand() % 10;
  }
}

// Print how two arrays `a` and `b` differ, up to some length `length`
// Returns true if the arrays are equal, and false otherwise
bool print_array_equality(num_t *a, num_t *b, size_t length) {
  bool are_equal = true;

  for (size_t i = 0; i < length; i++) {
    if (a[i] != b[i]) {
      are_equal = false;
      printf(
          "Arrays differ at index %ld, with values %d and %d\n", i, a[i], b[i]);
    }
  }

  return are_equal;
}

int main() {
  // Set up the input to scan
  num_t *input = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);
  fill_random_array(input, ARRAY_SIZE);

  // Set up the output for sequential function, to be used as ground truth for
  // comparison
  num_t *truth_output = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);

  // Set up the output for the parallel function
  num_t *output = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);

  // Run the sequential scan
  // TODO: Time this operation
  sequential_scan(input, truth_output, ARRAY_SIZE);

  // Run the parallel scan
  // TODO: Time this operation
  scan(input, output, ARRAY_SIZE);

  // Compare solutions
  bool are_equal = print_array_equality(truth_output, output, ARRAY_SIZE);

  if (are_equal) {
    printf("Success!\n");
  }

  return are_equal ? 0 : 1;
}

