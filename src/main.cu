#include "hip/hip_runtime.h"
#include <stdio.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define CUDA_ERROR(err, message) \
  do { \
    hipError_t err2 = err; \
    if (err2 != hipSuccess) { \
      fprintf(stderr, "%s: %s\n", message, hipGetErrorString(err2)); \
      exit(1); \
    } \
  } while (0);

#define GLOBAL_INDEX blockIdx.x * blockDim.x + threadIdx.x

#define BLOCK_SIZE ((size_t)128)

// `#define`s for addessing shared memory bank conflicts
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define OFFSET_ARRAY_INDEX(index) \
  ((index) >> NUM_BANKS + (index) >> (2 * LOG_NUM_BANKS))

// The size of the array to test on
static const size_t ARRAY_SIZE = 10000000;

typedef int32_t num_t;

// Performs exclusive Blelloch scan on a block level
// Also stores the sum of a total block in the `g_block_ends`
__global__
void blelloch_block_scan(
    const num_t *g_input, num_t *g_output, num_t *g_block_ends, size_t length) {
  __shared__ num_t s_temp[BLOCK_SIZE * 2];
  size_t global_index = GLOBAL_INDEX;
  size_t index = threadIdx.x;
  size_t offset = 1;

  // Copy global memory into shared
  if (global_index * 2 < length) {
    size_t i = global_index * 2;
    s_temp[index * 2] = g_input[i + OFFSET_ARRAY_INDEX(i)];
  }
  if (global_index * 2 + 1 < length) {
    size_t i = global_index * 2 + 1;
    s_temp[index * 2 + 1] = g_input[i + OFFSET_ARRAY_INDEX(i)];
  }

  // Up sweep
  for (size_t d = BLOCK_SIZE; d > 0; d /= 2) {
    __syncthreads();

    if (index < d) {
      size_t a = offset * (2 * index + 1) - 1;
      size_t b = offset * (2 * index + 2) - 1;
      a += OFFSET_ARRAY_INDEX(a);
      b += OFFSET_ARRAY_INDEX(b);
      s_temp[b] += s_temp[a];
    }

    offset *= 2;
  }

  // Reset last element
  if (index == 0) {
    size_t i = BLOCK_SIZE * 2 - 1;
    i += OFFSET_ARRAY_INDEX(i);
    // Save the block end
    if (g_block_ends != NULL) {
      g_block_ends[global_index / BLOCK_SIZE] = s_temp[i];
    }

    s_temp[i] = 0;
  }

  // Down sweep
  for (size_t d = 1; d < BLOCK_SIZE * 2; d *= 2) {
    offset /= 2;
    __syncthreads();
    if (index < d) {
      size_t a = offset * (2 * index + 1) - 1;
      size_t b = offset * (2 * index + 2) - 1;
      a += OFFSET_ARRAY_INDEX(a);
      b += OFFSET_ARRAY_INDEX(b);
      num_t t = s_temp[a];
      s_temp[a] = s_temp[b];
      s_temp[b] += t;
    }
  }

  __syncthreads();

  // Copy results into global memory
  if (global_index * 2 < length) {
    size_t i = index * 2;
    g_output[global_index * 2] = s_temp[i + OFFSET_ARRAY_INDEX(i)];
  }
  if (global_index * 2 + 1 < length) {
    size_t i = index * 2 + 1;
    g_output[global_index * 2 + 1] = s_temp[i + OFFSET_ARRAY_INDEX(i)];
  }
}

// Add the block scan ends back onto the original array
__global__
void add_block_scan_ends(
    num_t *g_input, const num_t *g_block_ends, size_t length) {

  size_t global_index = GLOBAL_INDEX;
  if (global_index < length) {
    g_input[global_index] += g_block_ends[global_index / (BLOCK_SIZE * 2)];
  }
}

// Perform level 1 scan on individual blocks of size `BLOCK_SIZE * 2`
void level1_scan(
    const num_t *g_input,
    num_t *g_output,
    const size_t length,
    num_t *g_block_ends,
    const size_t num_blocks) {
  blelloch_block_scan<<<num_blocks, BLOCK_SIZE>>>(
      g_input, g_output, g_block_ends, length);
  CUDA_ERROR(hipGetLastError(), "Couldn't perform block scan");
}

// Perform level 2 scan on groups of blocks where blocks are of size
// `BLOCK_SIZE * 2` and the groups are of size `BLOCK_SIZE * 2`
void level2_scan(
    const num_t *g_input,
    num_t *g_output,
    const size_t length,
    num_t *g_block_ends,
    const size_t num_blocks) {

  // Perform level 1 scan first
  level1_scan(g_input, g_output, length, g_block_ends, num_blocks);

  // Perform prefix sum of block scan ends
  size_t ends_num_blocks = 1 + (length - 1) / (BLOCK_SIZE * BLOCK_SIZE);
  blelloch_block_scan<<<ends_num_blocks, BLOCK_SIZE>>>(
      g_block_ends, g_block_ends, NULL, num_blocks);
  CUDA_ERROR(
      hipGetLastError(), "Couldn't perform block scan on block ends");

  // Add the block ends to the output
  add_block_scan_ends<<<num_blocks * 2, BLOCK_SIZE>>>(
      g_output, g_block_ends, length);
  CUDA_ERROR(hipGetLastError(), "Couldn't add block scan ends for level 2");
}

// Perform level 3 scan on groups of groups of blocks where each is of size
// `BLOCK_SIZE * 2`
void level3_scan(
    const num_t *g_input,
    num_t *g_output,
    const size_t length,
    num_t *g_block_ends,
    const size_t num_blocks,
    num_t *g_block_ends_ends,
    const size_t ends_num_blocks) {

  // Perform level 1 scan first
  level1_scan(g_input, g_output, length, g_block_ends, num_blocks);

  // Perform full level 2 scan on block ends
  level2_scan(
      g_block_ends, g_block_ends, num_blocks,
      g_block_ends_ends, ends_num_blocks);

  // Add the block ends to the output
  add_block_scan_ends<<<num_blocks * 2, BLOCK_SIZE>>>(
      g_output, g_block_ends, length);
  CUDA_ERROR(hipGetLastError(), "Couldn't add block scan ends for level 3");
}

// Performs all prefix sum on `input` and stores the result in `output` in
// parallel on a GPU
// Assumes both `input` and `output` are allocated with size `length`
// Returns the time it took to run the scan
double scan(const num_t *input, num_t *output, const size_t length) {
  hipError_t err;
  size_t array_size = sizeof(num_t) * length;
  size_t num_blocks = 1 + (length - 1) / (BLOCK_SIZE * 2);
  size_t ends_num_blocks = 1 + (length - 1) / (BLOCK_SIZE * BLOCK_SIZE);

  // Set up input on device
  num_t *g_input = NULL;
  err = hipMalloc((void **)&g_input, array_size);
  CUDA_ERROR(err, "Couldn't allocate memory for input on device");
  err = hipMemcpy(g_input, input, array_size, hipMemcpyHostToDevice);
  CUDA_ERROR(err, "Couldn't copy input to device");

  // Setup output on device
  num_t *g_output = NULL;
  err = hipMalloc((void **)&g_output, array_size);
  CUDA_ERROR(err, "Couldn't allocate memory for output on device");

  // Create array for block ends
  num_t *g_block_ends = NULL;
  err = hipMalloc((void**)&g_block_ends, sizeof(num_t) * num_blocks);
  CUDA_ERROR(err, "Couldn't allocated memory for scan_ends");

  // Create array for the ends of block ends
  num_t *g_block_ends_ends = NULL;
  err = hipMalloc((void**)&g_block_ends_ends, sizeof(num_t) * ends_num_blocks);
  CUDA_ERROR(err, "Couldn't allocated memory for scan_ends");

  // Setup timing kernels
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // Perform the scan
  if (length <= BLOCK_SIZE * 2) {
    level1_scan(g_input, g_output, length, NULL, num_blocks);
  } else if (length <= BLOCK_SIZE * BLOCK_SIZE * 4) {
    level2_scan(g_input, g_output, length, g_block_ends, num_blocks);
  } else if (length <= BLOCK_SIZE * BLOCK_SIZE * BLOCK_SIZE * 8) {
    level3_scan(
        g_input, g_output, length,
        g_block_ends, num_blocks,
        g_block_ends_ends, ends_num_blocks);
  } else {
    fprintf(stderr, "Couldn't handle array of size %ld\n", length);
    exit(1);
  }

  // Stop timing kernels
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed_time_ms;
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Copy results to host
  err = hipMemcpy(output, g_output, array_size, hipMemcpyDeviceToHost);
  CUDA_ERROR(err, "Couldn't copy output to host");

  // Free device allocated memory
  err = hipFree(g_input);
  CUDA_ERROR(err, "Couldn't free input on host");
  err = hipFree(g_output);
  CUDA_ERROR(err, "Couldn't free output on host");
  err = hipFree(g_block_ends);
  CUDA_ERROR(err, "Couldn't free block scan ends on host");

  return (double)elapsed_time_ms;
}

// Performs all prefix sum on `input` and stores the result in `output`
// sequentially on the CPU
// Assumes both `input` and `output` are allocated with size `length`
// Returns the time it took to run the scan
float sequential_scan(const num_t *input, num_t *output, size_t length) {
  // Start timer for sequential scan
  StopWatchInterface *sequential_timer = NULL;
  sdkCreateTimer(&sequential_timer);
  sdkStartTimer(&sequential_timer);

  for (size_t i = 1; i < length; i++) {
    output[i] = output[i - 1] + input[i - 1];
  }

  // Stop timers for sequential scan
  sdkStopTimer(&sequential_timer);
  return sdkGetTimerValue(&sequential_timer);
}

// Fills the array `array` with `length` random values from 0-9 inclusive
void fill_random_array(num_t *array, size_t length) {
  for (size_t i = 0; i < length; i++) {
    array[i] = rand() % 10;
  }
}

// Print how two arrays `a` and `b` differ, up to some length `length`
// Returns true if the arrays are equal, and false otherwise
bool print_array_equality(num_t *a, num_t *b, size_t length) {
  bool are_equal = true;

  for (size_t i = 0; i < length; i++) {
    if (a[i] != b[i]) {
      are_equal = false;
      printf(
          "Arrays differ at index %ld, with values %d and %d\n", i, a[i], b[i]);
    }
  }

  return are_equal;
}

int main() {
  // Set up the input to scan
  num_t *input = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);
  fill_random_array(input, ARRAY_SIZE);

  // Set up the output for sequential function, to be used as ground truth for
  // comparison
  num_t *truth_output = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);

  // Set up the output for the parallel function
  num_t *output = (num_t *)malloc(sizeof(num_t) * ARRAY_SIZE);

  // Run the sequential scan
  double sequential_time_elapsed_ms = sequential_scan(
      input, truth_output, ARRAY_SIZE);

  // Run the parallel scan
  double parallel_time_elapsed_ms = scan(input, output, ARRAY_SIZE);

  // Compare solutions
  bool are_equal = print_array_equality(truth_output, output, ARRAY_SIZE);

  if (are_equal) {
    printf("Success!\n");
  }

  printf("Sequential time: %f ms\n", sequential_time_elapsed_ms);
  printf("Parallel time: %f ms\n", parallel_time_elapsed_ms);
  printf(
      "Speed up: %f times\n",
      sequential_time_elapsed_ms / parallel_time_elapsed_ms);

  free(input);
  free(truth_output);
  free(output);

  return are_equal ? 0 : 1;
}

